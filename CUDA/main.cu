
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>

#define BLOCK_SIZE 512

__device__ double polynominal(double x){
    return 5*pow(x,4) + 4*pow(x,3) + x - 10*pow(x,2);
}

__global__ void  calculate(double* result, double start, double dx, long long int length){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;;

    double x = start + (float(idx) * dx);

    if(idx < length){
        result[idx] = (polynominal(x) + polynominal(x + dx)) * dx / 2.0f;
    }
}

__host__ double integral(double start, double stop, double dx) {
    double sum = 0;
    int length = (int)((stop - start) / dx);
    int size = length * sizeof(double);

    double* hostData = (double*)malloc(size);
    double* deviceData;
    hipMalloc((void**)&deviceData, size);

    int blocksCount = length / BLOCK_SIZE;

    if((length % BLOCK_SIZE) > 0) {
        blocksCount++;
    }

    calculate<<<blocksCount, BLOCK_SIZE>>>(deviceData, start, dx, length);

    hipMemcpy(hostData, deviceData, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < length; i++) {
        sum += hostData[i];
    }

    free(hostData);
    hipFree(deviceData);
    return sum;
}



int main() {
    double result = integral(0, 2, 0.001);

    printf("%f\n", result);
}